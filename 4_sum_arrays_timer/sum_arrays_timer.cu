#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <vector>
#include "freshman.h"

void sumArrays(float * a,float * b,float * res,const int size)
{
  for(int i=0;i<size;i+=4)
  {
    res[i]=a[i]+b[i];
    res[i+1]=a[i+1]+b[i+1];
    res[i+2]=a[i+2]+b[i+2];
    res[i+3]=a[i+3]+b[i+3];
  }
}

__global__ void sum_arrays(const float *a, const float *b, float *c, float n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {

    // set up device
    utills::initDevice(0);
    
    int nElem = 1 << 24 ;
    int nByte = nElem * sizeof(float);

    // malloc host memory
    float *h_a = (float *)malloc(nByte);
    float *h_b = (float *)malloc(nByte);
    float *h_res = (float *)malloc(nByte);
    float *h_res_from_gpu = (float *)malloc(nByte);

    memset(h_res, 0, nByte);
    memset(h_res_from_gpu, 0, nByte);

    // malloc device memory
    float *d_a, *d_b, *d_c;
    CHECK(hipMalloc((float**)&d_a, nByte));
    CHECK(hipMalloc((float**)&d_b, nByte));
    CHECK(hipMalloc((float**)&d_c, nByte));

    utills::initialData(h_a, nElem);
    utills::initialData(h_b, nElem);

    // copy data from host to device
    CHECK(hipMemcpy(d_a, h_a, nByte, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, h_b, nByte, hipMemcpyHostToDevice));

    CudaTimer timer;
    std::vector<int> blockSizes = {256, 512, 1024};

    for (int i = 0; i < blockSizes.size(); i++) {
      dim3 block(blockSizes[i], 1, 1);
      dim3 grid((nElem + block.x - 1) / block.x, 1, 1);
      
      timer.start();
      sum_arrays<<<grid, block>>>(d_a, d_b, d_c, nElem);
      timer.stop();
      printf("Data size: %d  <<<%d, %d>>> \n", nElem, grid.x, block.x);
      printf("Elapsed time: [%f] ms\n", timer.elapsed());
    }

    // copy data from device to host
    CHECK(hipMemcpy(h_res_from_gpu, d_c, nByte, hipMemcpyDeviceToHost));

    // free memory
    CHECK(hipFree(d_a));
    CHECK(hipFree(d_b));
    CHECK(hipFree(d_c));
    free(h_a);
    free(h_b);
    free(h_res);
    free(h_res_from_gpu);

    return 0;
}