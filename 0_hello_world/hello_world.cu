
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void hello_world(void) {
    printf("GPU: Hello world!\n");
}

int main() {
    printf("GPU: Hello world!\n");
    hello_world<<<1,10>>>();
    // Wait for GPU to finish
    hipDeviceSynchronize();
    hipDeviceReset();
    printf("GPU: Hello world finished!\n");
    return 0;
}